#include "world.cuh"

#include "vec.hpp"
#include <cstdint>

World::World(const Vec3<int> &size)
    : size(size), size_slice(size.x * size.y), size_grid(size_slice * size.z) {

  // alloc material attributes
  const size_t no_bytes_material_attributes = NO_MATERIALS * sizeof(float);

  hipMalloc(&material_attributes.courants, no_bytes_material_attributes);
  hipMalloc(&material_attributes.courants_squared, no_bytes_material_attributes);
  hipMalloc(&material_attributes.acoustic_impedances_doubled, no_bytes_material_attributes);

  hipMemset(material_attributes.courants, 0, no_bytes_material_attributes);
  hipMemset(material_attributes.courants_squared, 0, no_bytes_material_attributes);
  hipMemset(material_attributes.acoustic_impedances_doubled, 0, no_bytes_material_attributes);

  // alloc grid
  hipMalloc(&grid.material_ids, size_grid * sizeof(uint8_t));

  const size_t no_bytes_grid = size_grid * sizeof(float);

  hipMalloc(&grid.t0, no_bytes_grid);
  hipMalloc(&grid.t1, no_bytes_grid);
  hipMalloc(&grid.t2, no_bytes_grid);

  hipMemset(grid.t0, 0, no_bytes_grid);
  hipMemset(grid.t1, 0, no_bytes_grid);
  hipMemset(grid.t2, 0, no_bytes_grid);
}

World::~World() {
  // free material attributes
  hipFree(material_attributes.courants);
  hipFree(material_attributes.courants_squared);
  hipFree(material_attributes.acoustic_impedances_doubled);

  // free grid
  hipFree(grid.material_ids);
  hipFree(grid.t0);
  hipFree(grid.t1);
  hipFree(grid.t2);
}
