#include "hip/hip_runtime.h"
#include "world.cuh"

#include "cuda_utils.cuh"
#include <cstdlib>

namespace ears {

World::World(const Vec3i &size, const float courant, const dim3 dim_grid, const dim3 dim_block)
    : size(size), size_xy(size.x * size.y), size_xyz(size_xy * size.z), courant(courant),
      dim_grid(dim_grid), dim_block(dim_block) {

  const size_t no_bytes = size_xyz * sizeof(float);

  CUDA_CHECK(hipMalloc(&t0, no_bytes));
  CUDA_CHECK(hipMalloc(&t1, no_bytes));
  CUDA_CHECK(hipMalloc(&t2, no_bytes));

  CUDA_CHECK(hipMemset(t0, 0, no_bytes));
  CUDA_CHECK(hipMemset(t1, 0, no_bytes));
  CUDA_CHECK(hipMemset(t2, 0, no_bytes));
}

World::~World() {
  CUDA_CHECK(hipFree(t0));
  CUDA_CHECK(hipFree(t1));
  CUDA_CHECK(hipFree(t2));
}

const Vec3i &World::get_size() const {
  return size;
}

float World::get_courant() const {
  return courant;
}

#define GENERATE_WORLD_GET(time, dtype)                                                            \
  dtype World::get_##time(const Vec3i &pos) const {                                                \
    const int i = pos.x + pos.y * size.x + pos.z * size_xy;                                        \
    dtype val;                                                                                     \
    CUDA_CHECK(hipMemcpy(&val, time + i, sizeof(dtype), hipMemcpyDeviceToHost));                 \
    return val;                                                                                    \
  }

GENERATE_WORLD_GET(t0, float)
GENERATE_WORLD_GET(t1, float)
GENERATE_WORLD_GET(t2, float)

#define GENERATE_WORLD_SET(time, dtype)                                                            \
  void World::set_##time(const Vec3i &pos, const dtype val) const {                                \
    const int i = pos.x + pos.y * size.x + pos.z * size_xy;                                        \
    CUDA_CHECK(hipMemcpy(time + i, &val, sizeof(dtype), hipMemcpyHostToDevice));                 \
  }

GENERATE_WORLD_SET(t0, float)
GENERATE_WORLD_SET(t1, float)
GENERATE_WORLD_SET(t2, float)

__global__ void fdtd(const Vec3i size, const int size_xy, const float courant, float *const t0,
                     const float *const t1, const float *const t2) {
}

void World::step() {
  float *const swap = t2;
  t2 = t1;
  t1 = t0;
  t0 = swap;

  fdtd<<<dim_grid, dim_block>>>(size, size_xy, courant, t0, t1, t2);
  CUDA_CHECK(hipDeviceSynchronize());
}

void World::step(const int no_iter) {
  for (int i = 0; i < no_iter; i++)
    step();
}

} // namespace ears
