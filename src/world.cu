#include "world.hpp"

#include <cstdint>

World::World(const Vec3<int> &size)
    : size(size), slice_size(size.x * size.y), data_size(slice_size * size.z) {

  materials = new uint8_t[data_size]();
  data_t0 = new float[data_size]();
  data_t1 = new float[data_size]();
  data_t2 = new float[data_size]();
}

World::~World() {
  if (on_gpu) {
    hipFree(materials);
    hipFree(data_t0);
    hipFree(data_t1);
    hipFree(data_t2);
  } else {
    delete[] materials;
    delete[] data_t0;
    delete[] data_t1;
    delete[] data_t2;
  }
}

void World::to_gpu() {
  if (on_gpu)
    return;

  uint8_t *gpu_materials = nullptr;
  float *gpu_data_t0 = nullptr, *gpu_data_t1 = nullptr, *gpu_data_t2 = nullptr;

  hipMalloc(&gpu_materials, data_size * sizeof(uint8_t));
  hipMalloc(&gpu_data_t0, data_size * sizeof(float));
  hipMalloc(&gpu_data_t1, data_size * sizeof(float));
  hipMalloc(&gpu_data_t2, data_size * sizeof(float));

  hipMemcpy(gpu_materials, materials, data_size * sizeof(uint8_t), hipMemcpyHostToDevice);
  hipMemcpy(gpu_data_t0, data_t0, data_size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(gpu_data_t1, data_t1, data_size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(gpu_data_t2, data_t2, data_size * sizeof(float), hipMemcpyHostToDevice);

  delete[] materials;
  delete[] data_t0;
  delete[] data_t1;
  delete[] data_t2;

  materials = gpu_materials;
  data_t0 = gpu_data_t0;
  data_t1 = gpu_data_t1;
  data_t2 = gpu_data_t2;

  on_gpu = true;
}
