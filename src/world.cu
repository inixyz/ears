#include "hip/hip_runtime.h"
#include "world.cuh"

#include "fdtd.cuh"
#include "vec.hpp"
#include <algorithm>
#include <cmath>
#include <cstdint>

World::World(const Vec3i &size, const float grid_spacing_distance)
    : size(size), size_slice(size.x * size.y), size_grid(size_slice * size.z),
      grid_spacing_distance(grid_spacing_distance),
      dim_grid(dim3(size.x / dim_block.x, size.y / dim_block.y, size.z / dim_block.z)) {

  // alloc material attributes
  const size_t no_bytes_material_attributes = NO_MATERIALS * sizeof(float);

  hipMalloc(&material_attributes.courants, no_bytes_material_attributes);
  hipMalloc(&material_attributes.courants_squared, no_bytes_material_attributes);
  hipMalloc(&material_attributes.acoustic_impedances_doubled, no_bytes_material_attributes);

  hipMemset(material_attributes.courants, 0, no_bytes_material_attributes);
  hipMemset(material_attributes.courants_squared, 0, no_bytes_material_attributes);
  hipMemset(material_attributes.acoustic_impedances_doubled, 0, no_bytes_material_attributes);

  // alloc grid
  hipMalloc(&grid.material_id, size_grid * sizeof(uint8_t));

  const size_t no_bytes_grid = size_grid * sizeof(float);

  hipMalloc(&grid.t0, no_bytes_grid);
  hipMalloc(&grid.t1, no_bytes_grid);
  hipMalloc(&grid.t2, no_bytes_grid);

  hipMemset(grid.t0, 0, no_bytes_grid);
  hipMemset(grid.t1, 0, no_bytes_grid);
  hipMemset(grid.t2, 0, no_bytes_grid);
}

World::~World() {
  // free material attributes
  hipFree(material_attributes.courants);
  hipFree(material_attributes.courants_squared);
  hipFree(material_attributes.acoustic_impedances_doubled);

  // free grid
  hipFree(grid.material_id);
  hipFree(grid.t0);
  hipFree(grid.t1);
  hipFree(grid.t2);
}

const Vec3i &World::get_size() const {
  return size;
}

#define GENERATE_WORLD_GET(grid_member, dtype)                                                     \
  dtype World::get_##grid_member(const Vec3i &pos) const {                                         \
    const int i = pos.x + pos.y * size.x + pos.z * size_slice;                                     \
    dtype val;                                                                                     \
    hipMemcpy(&val, grid.grid_member + i, sizeof(dtype), hipMemcpyDeviceToHost);                 \
    return val;                                                                                    \
  }

GENERATE_WORLD_GET(material_id, uint8_t)
GENERATE_WORLD_GET(t0, float)
GENERATE_WORLD_GET(t1, float)
GENERATE_WORLD_GET(t2, float)

#define GENERATE_WORLD_SET(grid_member, dtype)                                                     \
  void World::set_##grid_member(const Vec3i &pos, const dtype val) const {                         \
    const int i = pos.x + pos.y * size.x + pos.z * size_slice;                                     \
    hipMemcpy(grid.grid_member + i, &val, sizeof(dtype), hipMemcpyHostToDevice);                 \
  }

GENERATE_WORLD_SET(material_id, uint8_t)
GENERATE_WORLD_SET(t0, float)
GENERATE_WORLD_SET(t1, float)
GENERATE_WORLD_SET(t2, float)

void World::compute_material_attributes() const {
  // compute max sound speed found in materials
  auto cmp_sound_speed = [](const Material lhs, const Material rhs) {
    return lhs.sound_speed < rhs.sound_speed;
  };
  const float max_sound_speed =
      std::max_element(materials.begin(), materials.end(), cmp_sound_speed)->sound_speed;

  const float grid_spacing_temporal = grid_spacing_distance / (max_sound_speed * std::sqrt(3));

  for (int i = 0; i < NO_MATERIALS; i++) {
    const float courant = materials[i].sound_speed * grid_spacing_temporal / grid_spacing_distance;
    const float courant_squared = courant * courant;
    const float acoustic_impedance_doubled = 2 * materials[i].acoustic_impedance;

    hipMemcpy(material_attributes.courants + i, &courant, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(material_attributes.courants_squared + i, &courant_squared, sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(material_attributes.acoustic_impedances_doubled + i, &acoustic_impedance_doubled,
               sizeof(float), hipMemcpyHostToDevice);
  }
}

void World::step(const int no_iterations) {
  for (int i = 0; i < no_iterations; i++) {
    // advance time axis
    std::swap(grid.t1, grid.t0);
    std::swap(grid.t2, grid.t0);

    fdtd_step<<<dim_grid, dim_block>>>(size, size_slice, material_attributes, grid);
    hipDeviceSynchronize();
  }
}
